#include "hip/hip_runtime.h"
// Copyright (C) 2019-2022 Aleo Systems Inc.
// This file is part of the snarkVM library.

// The snarkVM library is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// The snarkVM library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with the snarkVM library. If not, see <https://www.gnu.org/licenses/>.

#include <hip/hip_runtime.h>
#include <chrono>
#include "snarkvm.cu"

#include <iostream>

#include <queue>
#include <memory>
#include <mutex>
#include <condition_variable>


#ifndef __CUDA_ARCH__

template<typename T>
class threadsafe_queue {
private:
    mutable std::mutex mut;
    std::queue<T> data_queue;
    std::condition_variable data_cond;
public:
    threadsafe_queue(){}
    threadsafe_queue(threadsafe_queue const& other) {
        std::lock_guard<std::mutex> lk(other.mut);
        data_queue=other.data_queue;
    }

    void push(T new_value) {
        std::lock_guard<std::mutex> lk(mut);
        data_queue.push(new_value);
        data_cond.notify_one();
    }

    void wait_and_pop(T& value) {
        std::unique_lock<std::mutex> lk(mut);
        data_cond.wait(lk,[this]{return !data_queue.empty();});
        value=data_queue.front();
        data_queue.pop();
    }

    std::shared_ptr<T> wait_and_pop() {
        std::unique_lock<std::mutex> lk(mut);
        data_cond.wait(lk,[this]{return !data_queue.empty();});
        std::shared_ptr<T> res(std::make_shared<T>(data_queue.front()));
        data_queue.pop();
        return res;
    }

    bool try_pop(T& value) {
        std::lock_guard<std::mutex> lk(mut);
        if(data_queue.empty())
            return false;
        value=data_queue.front();
        data_queue.pop();
        return true;
    }

    std::shared_ptr<T> try_pop() {
        std::lock_guard<std::mutex> lk(mut);
        if(data_queue.empty())
            return std::shared_ptr<T>();
        std::shared_ptr<T> res(std::make_shared<T>(data_queue.front()));
        data_queue.pop();
        return res;
    }

    bool empty() const {
        std::lock_guard<std::mutex> lk(mut);
        return data_queue.empty();
    }
};

// Lazy instantiation of snarkvm_t
class snarkvm_singleton_t {
    bool failed = false;
    int  iindex  = 0;
    snarkvm_t *snarkvm = nullptr;

public:
    snarkvm_singleton_t(int ii) {
        iindex = ii;
    }
    ~snarkvm_singleton_t() {
        delete snarkvm;
        snarkvm = nullptr;
    }
    bool ok() {
        if (!failed && snarkvm == nullptr) {
            // SNP TODO: max domain size?
            snarkvm = new snarkvm_t(17);
            if (snarkvm == nullptr) {
                failed = true;
            }
        }
        cout << "-----------------------vm ok----------------------";

        return snarkvm != nullptr;
    }
    snarkvm_t* get() {
        assert (ok());
        return snarkvm;
    }
    int get_index(){
        return iindex;
    }
};
//snarkvm_singleton_t snarkvm_g;

static threadsafe_queue<snarkvm_singleton_t*> snarkvm_g;
bool initCode()
{
    for (int i = 0; i < 32 /*!!*/; i++) {
        snarkvm_g.push(new snarkvm_singleton_t(i));
    }
    return true;
}

static bool bSzArrCountryCodeInit  = initCode();


#ifndef __CUDA_ARCH__

extern "C" {
RustError snarkvm_ntt(fr_t* inout, uint32_t lg_domain_size,
                      NTT::InputOutputOrder ntt_order, NTT::Direction ntt_direction,
                      NTT::Type ntt_type)
{

    std::shared_ptr<snarkvm_singleton_t*> p = snarkvm_g.wait_and_pop();

    RustError ret = RustError{hipErrorOutOfMemory};
    try{
        if ((*p)->ok()) {
            ret = (*p)->get()->NTT(inout, inout, lg_domain_size, ntt_order,
                                   ntt_direction, ntt_type);
            snarkvm_g.push((*p));
            return ret;
        }
        snarkvm_g.push((*p));
    }
    catch (...){
        snarkvm_g.push((*p));
    }
    return ret;

    //if (!snarkvm_g.ok()) {
    //    return RustError{hipErrorOutOfMemory};
    //}
    //return snarkvm_g->NTT(inout, inout, lg_domain_size, ntt_order,
    //                      ntt_direction, ntt_type);
}

RustError snarkvm_polymul(fr_t* out,
                          size_t pcount, fr_t** polynomials, size_t* plens,
                          size_t ecount, fr_t** evaluations, size_t* elens,
                          uint32_t lg_domain_size) {

    std::shared_ptr<snarkvm_singleton_t*> p = snarkvm_g.wait_and_pop();

    RustError ret = RustError{hipErrorOutOfMemory};
    try{
        if ((*p)->ok()) {

            ret = (*p)->get()->PolyMul(out,
                                       pcount, polynomials, plens,
                                       ecount, evaluations, elens,
                                       lg_domain_size);
            snarkvm_g.push((*p));
            return ret;
        }
        snarkvm_g.push((*p));
    }
    catch (...){
        snarkvm_g.push((*p));
    }
    return ret;


    //if (!snarkvm_g.ok()) {
    //    return RustError{hipErrorOutOfMemory};
    //}
    //return snarkvm_g->PolyMul(out,
    //                          pcount, polynomials, plens,
    //                          ecount, evaluations, elens,
    //                          lg_domain_size);
}

RustError snarkvm_msm(point_t* out, const affine_t points[], size_t npoints,
                      const scalar_t scalars[], size_t ffi_affine_size) {

    high_resolution_clock::time_point beginTime = high_resolution_clock::now();

    std::shared_ptr<snarkvm_singleton_t*> p = snarkvm_g.wait_and_pop();

    high_resolution_clock::time_point endTime = high_resolution_clock::now();

    milliseconds timeInterval = std::chrono::duration_cast<milliseconds>(endTime - beginTime);
    cout <<  "### snarkvm_msm wait_and_pop time " << timeInterval.count() << "ms\n";

    RustError ret = RustError{hipErrorOutOfMemory};
    try{
        if ((*p)->ok()) {
            cout << "vm index: " << (*p)->get_index()  << "\r\n";
            ret = (*p)->get()->MSM(out, points, npoints, scalars, ffi_affine_size);
            snarkvm_g.push((*p));
            return ret;
        }
        snarkvm_g.push((*p));
    }
    catch (...){
        snarkvm_g.push((*p));
    }
    return ret;

    //if (!snarkvm_g.ok()) {
    //    return RustError{hipErrorOutOfMemory};
    //}
    //return snarkvm_g->MSM(out, points, npoints, scalars, ffi_affine_size);
}
}
#endif // __CUDA_ARCH__

#endif
